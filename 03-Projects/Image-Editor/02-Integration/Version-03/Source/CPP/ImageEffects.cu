#include "hip/hip_runtime.h"
#include "ImageEffects.cuh"

//! Cuda Utils Function Definitions
bool CudaUtils::memAlloc(void **devPtr, size_t size)
{
    // Code
    hipError_t result = hipMalloc(devPtr, size);
    if (result != hipSuccess)
        return false;
    return true;
}

bool CudaUtils::memSet(void *devPtr, int value, size_t count)
{
    // Code
    hipError_t result = hipMemset(devPtr, value, count);
    if (result != hipSuccess)
        return false;
    return true;
}

bool CudaUtils::memCopy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    // Code
    hipError_t result = hipMemcpy(dst, src, count, kind);
    if (result != hipSuccess)
        return false;
    return true;
}

void CudaUtils::memFree(void **devPtr)
{
    // Code
    if (*devPtr)
    {
        if (hipFree(*devPtr) == hipSuccess);
            *devPtr = nullptr;
    }
}

void CudaUtils::convertImageToPixelArr(uchar_t *imageData, uchar3 *pixelArray, size_t size)
{
    // Code
    for (size_t i = 0; i < size; i++, imageData += 3)
    {
        pixelArray[i].x = imageData[2];
        pixelArray[i].y = imageData[1];
        pixelArray[i].z = imageData[0];
    }
}

void CudaUtils::convertPixelArrToImage(uchar3 *pixelArray, uchar_t *imageData, size_t size)
{
    // Code
    for (size_t i = 0; i < size; i++, imageData += 3)
    {
        imageData[2] = pixelArray[i].x;
        imageData[1] = pixelArray[i].y;
        imageData[0] = pixelArray[i].z;
    }
}

// Variable Declarations
    // cv::Mat cuda_sobel_input_image, cuda_sobel_output_image;

    // Code

    // Reading Input Image
    // cuda_sobel_input_image = cv::imread(cuda_sobel_input_file, cv::IMREAD_GRAYSCALE);
    // cuda_sobel_output_image = cuda_sobel_input_image.clone();

    // double result = sobel_operator(&cuda_sobel_input_image, &cuda_sobel_output_image);

    // cuda_sobel_output_image.convertTo(cuda_sobel_output_image, CV_8UC1);

    // cv::imwrite(cuda_sobel_output_file, cuda_sobel_output_image);

    // // Cleanup Code
    // sobel_cuda_cleanup();
    // cuda_sobel_output_image.release();
    // cuda_sobel_input_image.release();

//! Sobel Operator Function Definitions
SobelOperator::SobelOperator()
{
    device_input = nullptr;
    device_output = nullptr;
    host_kernel = nullptr;
    device_kernel = nullptr;

    cudaUtils = new CudaUtils();
}

SobelOperator::~SobelOperator()
{
    cudaUtils->memFree((void**)&device_kernel);
    cudaUtils->memFree((void**)&device_output);
    cudaUtils->memFree((void**)&device_input);

    if (host_kernel)
    {
        delete[] host_kernel;
        host_kernel = nullptr;
    }

    delete cudaUtils;
    cudaUtils = nullptr;
}

void SobelOperator::applySobelEdgeDetection(cv::Mat *input_image, cv::Mat *output_image)
{
    // Variable Declarations
    float kernel_sum = 0.0f;
    float sigma = 1.0f;

    // Code

    // Get Image Properties
    int image_width = input_image->cols;
    int image_height = input_image->rows;
    int image_size = image_height * image_width * sizeof(unsigned char);

    // Create Gaussian Kernel
    host_kernel = new float[CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE];
    int kernel_radius = CUDA_GAUSSIAN_KERNEL_SIZE / 2;

    for (int i = -kernel_radius; i <= kernel_radius; i++) 
    {
        for (int j = -kernel_radius; j <= kernel_radius; j++)
        {
            int index = (i + kernel_radius) * kernel_radius + (j + kernel_radius);
            host_kernel[index] = exp(-(i * i + j + j) / (2.0f * sigma * sigma));
            kernel_sum = kernel_sum + host_kernel[index];
        }
    }

    for (int i = 0; i < CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE; i++)
        host_kernel[i] = host_kernel[i] / kernel_sum;

    cudaUtils->memAlloc((void **)&device_input, image_size);
    cudaUtils->memAlloc((void **)&device_output, image_size);
    cudaUtils->memAlloc((void **)&device_kernel, CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE * sizeof(float));
    
    cudaUtils->memCopy(device_input, input_image->data, image_size, hipMemcpyHostToDevice);
    cudaUtils->memCopy(device_kernel, host_kernel, CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Kernel Configuration
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(image_height, image_width);

    // CUDA Kernel
    gaussianBlurKernel<<<dimGrid, dimBlock>>>(device_input, device_output, image_width, image_height, device_kernel);
    sobelFilterKernel<<<dimGrid, dimBlock>>>(device_input, device_output, input_image->cols, input_image->rows);

    cudaUtils->memCopy(output_image->data, device_output, image_size, hipMemcpyDeviceToHost);
}



// // Variable Declarations
// cv::Mat cuda_canny_input_image, cuda_canny_output_image;
// cv::String cuda_canny_input_file, cuda_canny_output_file;

// // Code
// cuda_canny_input_file = input_file;
// filesystem::path output_path = filesystem::path(input_file).filename();
// string output_file_name = output_path.string();

// #if (OS == 1)
//     cuda_canny_output_file = output_file + "\\Canny_CUDA_" + output_file_name;
// #elif (OS == 2)
//     cuda_canny_output_file = output_file + "/Canny_CUDA_" + output_file_name;
// #endif

// cuda_canny_input_image = cv::imread(cuda_canny_input_file, cv::IMREAD_GRAYSCALE);
// cuda_canny_output_image = cuda_canny_input_image.clone();

// double result = canny_operator(cuda_canny_input_image.data, cuda_canny_output_image.data, cuda_canny_input_image.cols, cuda_canny_input_image.rows);

// cuda_canny_output_image.convertTo(cuda_canny_output_image, CV_8UC1);

// cv::imwrite(cuda_canny_output_file, cuda_canny_output_image);

// canny_cuda_cleanup();
// cuda_canny_output_image.release();
// cuda_canny_input_image.release();

CannyOperator::CannyOperator()
{
    input_pixels = nullptr;
    output_pixels = nullptr;
    segment_pixels = nullptr, 
    final_result = nullptr;
    gradient_pixels = nullptr;
    max_pixels = nullptr;
    gaussian_kernel_gpu = nullptr;
    sobel_kernel_x_gpu = nullptr;
    sobel_kernel_y_gpu = nullptr;

    cudaUtils = new CudaUtils();
}

CannyOperator::~CannyOperator()
{
    cudaUtils->memFree((void**)&final_result);
    cudaUtils->memFree((void**)&sobel_kernel_y_gpu);
    cudaUtils->memFree((void**)&sobel_kernel_x_gpu);
    cudaUtils->memFree((void**)&gaussian_kernel_gpu);
    cudaUtils->memFree((void**)&segment_pixels);
    cudaUtils->memFree((void**)&max_pixels);
    cudaUtils->memFree((void**)&gradient_pixels);
    cudaUtils->memFree((void**)&output_pixels);
    cudaUtils->memFree((void**)&input_pixels);

    delete cudaUtils;
    cudaUtils = nullptr;
}

void CannyOperator::applyCannyEdgeDetection(uchar_t *input_image_data, uchar_t *output_image_data, int image_width, int image_height)
{
    // Variable Declarations
	const double gaussian_kernel[9] = 
    {
		1, 2, 1,
		2, 4, 2,
		1, 2, 1
	};

	const schar_t sobel_kernel_x[] = 
    {   
        -1, 0, 1,
		-2, 0, 2,
		-1, 0, 1 
    };

	const schar_t sobel_kernel_y[] = 
    {    
        1, 2, 1,
		0, 0, 0,
		-1,-2,-1 
    };

	const int NUM_BLOCKS = (image_height * image_width) / THREADS_PER_BLOCK;

	cudaUtils->memAlloc((void**)&input_pixels, sizeof(uchar_t) * image_height * image_width);
	cudaUtils->memAlloc((void**)&output_pixels, sizeof(uchar_t) * image_height * image_width);
	cudaUtils->memAlloc((void**)&gradient_pixels, sizeof(double) * image_height * image_width);
	cudaUtils->memAlloc((void**)&final_result, sizeof(uchar_t) * image_height * image_width);
	cudaUtils->memAlloc((void**)&max_pixels, sizeof(double) * image_height * image_width);
	cudaUtils->memAlloc((void**)&segment_pixels, sizeof(uchar_t) * image_height * image_width);
	cudaUtils->memAlloc((void**)&gaussian_kernel_gpu, sizeof(double) * CV_GAUSSIAN_KERNEL_SIZE * CV_GAUSSIAN_KERNEL_SIZE);
	cudaUtils->memAlloc((void**)&sobel_kernel_x_gpu, sizeof(schar_t) * 3 * 3);
	cudaUtils->memAlloc((void**)&sobel_kernel_y_gpu, sizeof(schar_t) * 3 * 3);

	cudaUtils->memCopy(input_pixels, input_image_data, image_height * image_width * sizeof(uchar_t), hipMemcpyHostToDevice);
	cudaUtils->memCopy(gaussian_kernel_gpu, gaussian_kernel, sizeof(double) * CV_GAUSSIAN_KERNEL_SIZE * CV_GAUSSIAN_KERNEL_SIZE, hipMemcpyHostToDevice);
	cudaUtils->memCopy(sobel_kernel_x_gpu, sobel_kernel_x, sizeof(schar_t) * CV_GAUSSIAN_KERNEL_SIZE * CV_GAUSSIAN_KERNEL_SIZE, hipMemcpyHostToDevice);
	cudaUtils->memCopy(sobel_kernel_y_gpu, sobel_kernel_y, sizeof(schar_t) * CV_GAUSSIAN_KERNEL_SIZE * CV_GAUSSIAN_KERNEL_SIZE, hipMemcpyHostToDevice);

    // CUDA Kernel Call
    gaussianBlur<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(input_pixels, output_pixels, image_width, image_height, gaussian_kernel_gpu);
    sobelFilter<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(gradient_pixels, segment_pixels, output_pixels, image_width, image_height, sobel_kernel_x_gpu, sobel_kernel_y_gpu);
    
    cudaUtils->memCopy(max_pixels, gradient_pixels, image_height * image_width * sizeof(double), hipMemcpyDeviceToDevice);
    
    nonMaxSuppression<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(max_pixels, gradient_pixels, segment_pixels, image_width, image_height);
    doubleThreshold<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(output_pixels, max_pixels, CUDA_THRESHOLD * CUDA_RATIO, CUDA_THRESHOLD, image_width, image_height);
    
    cudaUtils->memCopy(final_result, output_pixels, image_height * image_width * sizeof(uchar_t), hipMemcpyDeviceToDevice);
    
    edgeHysteresis<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(final_result, output_pixels, image_width, image_height);

	cudaUtils->memCopy(output_image_data, final_result, image_width * image_height * sizeof(uchar_t), hipMemcpyDeviceToHost);
}



// CUDA Kernels
__global__ void gaussianBlurKernel(
    unsigned char *cuda_sobel_input_image, 
    unsigned char *cuda_sobel_output_image, 
    int width, 
    int height, 
    float *kernel
)
{
    // Code
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        float blur_pixel = 0.0f;
        int kernel_radius = CUDA_GAUSSIAN_KERNEL_SIZE / 2;

        for (int i = -kernel_radius; i <= kernel_radius; i++)
        {
            for (int j = -kernel_radius; j <= kernel_radius; j++)
            {
                int x_offset = x + i;
                int y_offset = y + j;

                if (x_offset >= 0 && x_offset < width && y_offset >= 0 && y_offset < height)
                {
                    int input_index = y_offset * width + x_offset;
                    int kernel_index = (i + kernel_radius) * CUDA_GAUSSIAN_KERNEL_SIZE + (j + kernel_radius);
                    blur_pixel = blur_pixel + static_cast<float>(cuda_sobel_input_image[input_index]) * kernel[kernel_index];
                }
            }
        }

        cuda_sobel_output_image[y * width + x] = static_cast<unsigned char>(blur_pixel);
    }
}

__global__ void sobelFilterKernel(
    unsigned char *cuda_sobel_input_image, 
    unsigned char *cuda_sobel_output_image, 
    unsigned int image_width, 
    unsigned int image_height
)
{
    // Variable Declarations
    int sobel_x[CUDA_SOBEL_KERNEL_SIZE][CUDA_SOBEL_KERNEL_SIZE] = 
    {
        { -1, 0, 1 },
        { -2, 0, 2 },
        { -1, 0, 1 }
    };

    int sobel_y[CUDA_SOBEL_KERNEL_SIZE][CUDA_SOBEL_KERNEL_SIZE] = 
    {
        { -1, -2, -1 },
        {  0,  0,  0 },
        {  1,  2,  1 }
    };

    // Code
    int num_rows = blockIdx.x * blockDim.x + threadIdx.x;
    int num_columns = blockIdx.y * blockDim.y + threadIdx.y;

    int index = (num_rows * image_width) + num_columns;

    if ((num_columns < (image_width - 1)) && (num_rows < (image_height - 1)))
    {
        float gradient_x =  (cuda_sobel_input_image[index] * sobel_x[0][0]) + (cuda_sobel_input_image[index + 1] * sobel_x[0][1]) + (cuda_sobel_input_image[index + 2] * sobel_x[0][2]) +
                            (cuda_sobel_input_image[index] * sobel_x[1][0]) + (cuda_sobel_input_image[index + 1] * sobel_x[1][1]) + (cuda_sobel_input_image[index + 2] * sobel_x[1][2]) +
                            (cuda_sobel_input_image[index] * sobel_x[2][0]) + (cuda_sobel_input_image[index + 1] * sobel_x[2][1]) + (cuda_sobel_input_image[index + 2] * sobel_x[2][2]);

        float gradient_y =  (cuda_sobel_input_image[index] * sobel_y[0][0]) + (cuda_sobel_input_image[index + 1] * sobel_y[0][1]) + (cuda_sobel_input_image[index + 2] * sobel_y[0][2]) +
                            (cuda_sobel_input_image[index] * sobel_y[1][0]) + (cuda_sobel_input_image[index + 1] * sobel_y[1][1]) + (cuda_sobel_input_image[index + 2] * sobel_y[1][2]) +
                            (cuda_sobel_input_image[index] * sobel_y[2][0]) + (cuda_sobel_input_image[index + 1] * sobel_y[2][1]) + (cuda_sobel_input_image[index + 2] * sobel_y[2][2]);

        float gradient = sqrtf(gradient_x * gradient_x + gradient_y * gradient_y);

        if (gradient > 255)
            gradient = 255;

        if (gradient < 0)
            gradient = 0;

        __syncthreads();

        cuda_sobel_output_image[index] = gradient;
    }
}

__global__ void gaussianBlur(
    const uchar_t* input_data, 
    uchar_t* output_data, 
    int image_width, 
    int image_height, 
    double* gaussian_kernel
)
{
    // Code
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	const int offset_xy = ((CV_GAUSSIAN_KERNEL_SIZE - 1) / 2);

	if ((id >= 0 && id < image_height * image_width))
    {
        double kernelSum = 0;
        double blurPixel = 0;

        for (int i = 0; i < CV_GAUSSIAN_KERNEL_SIZE; i++) 
        {
            for (int j = 0; j < CV_GAUSSIAN_KERNEL_SIZE; j++) 
            {
                if (((id + ((i - offset_xy) * image_width) + j - offset_xy) >= 0) && 
                    ((id + ((i - offset_xy) * image_width) + j - offset_xy) <= image_height * image_width - 1) && 
                    (((id % image_width) + j - offset_xy) >= 0) && 
                    (((id % image_width) + j - offset_xy) <= (image_width - 1))) 
                    {
                        blurPixel = blurPixel + gaussian_kernel[i * CV_GAUSSIAN_KERNEL_SIZE + j] * input_data[id + ((i - offset_xy) * image_width) + j - offset_xy];
                        kernelSum = kernelSum + gaussian_kernel[i * CV_GAUSSIAN_KERNEL_SIZE + j];
                    }
            }
        }
        
        output_data[id] = (uchar_t)(blurPixel / kernelSum);
    }
}

__global__ void sobelFilter(
    double* gradient_pixels, 
    uchar_t* segment_pixels, 
    const uchar_t* input_data, 
    int image_width, 
    int image_height, 
    schar_t* sobel_kernel_x, 
    schar_t* sobel_kernel_y
) 
{
    // Code
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if ((id >= 0 && id < image_height * image_width))
    {
        int offset_xy = 1;
        double convolve_X = 0.0;
        double convolve_Y = 0.0;
        int k = 0;
        int segment = 0;

        int x = id % image_width;
        int y = id / image_width;
        
        if (x < offset_xy || x >= image_width - offset_xy || y < offset_xy || y >= image_height - offset_xy)
            return;
        
        int src_id = x + (y * image_width);

        for (int ky = -offset_xy; ky <= offset_xy; ky++) {
            for (int kx = -offset_xy; kx <= offset_xy; kx++) {
                convolve_X += input_data[src_id + (kx + (ky * image_width))] * sobel_kernel_x[k];
                convolve_Y += input_data[src_id + (kx + (ky * image_width))] * sobel_kernel_y[k];
                k++;
            }
        }

        if (convolve_X == 0.0 || convolve_Y == 0.0) 
        {
            gradient_pixels[src_id] = 0;
        }
        else 
        {
            gradient_pixels[src_id] = ((std::sqrt((convolve_X * convolve_X) + (convolve_Y * convolve_Y))));
            double theta = std::atan2(convolve_Y, convolve_X);
            theta = theta * (360.0 / (2.0 * M_PI));

            if ((theta <= 22.5 && theta >= -22.5) || (theta <= -157.5) || (theta >= 157.5))
                segment = 1;
            else if ((theta > 22.5 && theta <= 67.5) || (theta > -157.5 && theta <= -112.5))
                segment = 2;
            else if ((theta > 67.5 && theta <= 112.5) || (theta >= -112.5 && theta < -67.5))
                segment = 3;
            else if ((theta >= -67.5 && theta < -22.5) || (theta > 112.5 && theta < 157.5))
                segment = 4;
        }

        segment_pixels[src_id] = (uchar_t)segment;
    }
}

__global__ void nonMaxSuppression(
    double* max_pixels, 
    double* gradient_pixels, 
    uchar_t* segment_pixels, 
    int image_width, 
    int image_height
) 
{
    // Code
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if ((id >= 0 && id < image_height * image_width))
    {
        switch (segment_pixels[id]) 
        {
            case 1:
                if (segment_pixels[id - 1] >= gradient_pixels[id] || gradient_pixels[id + 1] > gradient_pixels[id])
                    max_pixels[id] = 0;
            break;

            case 2:
                if (gradient_pixels[id - (image_width - 1)] >= gradient_pixels[id] || gradient_pixels[id + (image_width - 1)] > gradient_pixels[id])
                    max_pixels[id] = 0;
            break;

            case 3:
                if (gradient_pixels[id - (image_width)] >= gradient_pixels[id] || gradient_pixels[id + (image_width)] > gradient_pixels[id])
                    max_pixels[id] = 0;
            break;

            case 4:
                if (gradient_pixels[id - (image_width + 1)] >= gradient_pixels[id] || gradient_pixels[id + (image_width + 1)] > gradient_pixels[id])
                    max_pixels[id] = 0;
            break;

            default:
                max_pixels[id] = 0;
            break;
        }
    }
}

__global__ void doubleThreshold(
    uchar_t* out, 
    double* max_pixels, 
    int strong_threshold, 
    int weak_threshold,
    int image_width, 
    int image_height
) 
{
    // Code
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if ((id >= 0 && id < image_height * image_width))
    {
        if (max_pixels[id] > strong_threshold)
		    out[id] = 255;
        else if (max_pixels[id] > weak_threshold)
            out[id] = 100;
        else
            out[id] = 0;
    }
}

__global__ void edgeHysteresis(
    uchar_t* out, 
    uchar_t* in, 
    int image_width, 
    int image_height
) 
{
    // Code
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if ((id >= 0 && id < image_height * image_width))
    {
        if (in[id] == 100) 
        {
            if (in[id - 1] == 255 || in[id + 1] == 255 ||
                in[id - image_width] == 255 || in[id + image_width] == 255 ||
                in[id - image_width - 1] == 255 || in[id - image_width + 1] == 255 ||
                in[id + image_width - 1] == 255 || in[id + image_width + 1] == 255)
                {
                    out[id] = 255;
                }
                
            else
                out[id] = 0;
        }
	}
}
