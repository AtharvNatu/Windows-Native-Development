#include "hip/hip_runtime.h"
#include "ImageEffects.cuh"

//! Cuda Utils Function Definitions
bool CudaUtils::memAlloc(void **devPtr, size_t size)
{
    // Code
    hipError_t result = hipMalloc(devPtr, size);
    if (result != hipSuccess)
        return false;
    return true;
}

bool CudaUtils::memSet(void *devPtr, int value, size_t count)
{
    // Code
    hipError_t result = hipMemset(devPtr, value, count);
    if (result != hipSuccess)
        return false;
    return true;
}

bool CudaUtils::memCopy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    // Code
    hipError_t result = hipMemcpy(dst, src, count, kind);
    if (result != hipSuccess)
        return false;
    return true;
}

void CudaUtils::memFree(void **devPtr)
{
    // Code
    if (*devPtr)
    {
        if (hipFree(*devPtr) == hipSuccess)
            *devPtr = nullptr;
    }
}

__global__ void cudaDesaturation(uchar3* inputData, uchar3* outputData, size_t imageSize)
{
    // Code
    int pixelId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (pixelId < imageSize)
    {
        uchar grayscale = (uchar)(
                        (0.3f * (uchar)inputData[pixelId].z) +
                        (0.59f * (uchar)inputData[pixelId].y) +
                        (0.11f * (uchar)inputData[pixelId].x)
                    );
        
        outputData[pixelId].x = grayscale;
        outputData[pixelId].y = grayscale;
        outputData[pixelId].z = grayscale;
    }
}

__global__ void cudaDesaturation(unsigned char* inputData, unsigned char* outputData, int imageWidth, int imageHeight)
{
    // Code
    int xColumn = blockIdx.x * blockDim.x + threadIdx.x;
    int yRow = blockIdx.y * blockDim.y + threadIdx.y;

    if (xColumn < imageWidth && yRow < imageHeight)
    {
        int pixelId = (yRow * imageWidth + xColumn) * 3;

        uchar grayscale = (uchar)(
                        (0.3f * (uchar)inputData[pixelId + 0]) +
                        (0.59f * (uchar)inputData[pixelId + 1]) +
                        (0.11f * (uchar)inputData[pixelId + 2])
                    );
        
        outputData[pixelId + 0] = grayscale;
        outputData[pixelId + 1] = grayscale;
        outputData[pixelId + 2] = grayscale;
    }
}

__global__ void cudaSepia(unsigned char* inputData, unsigned char* outputData, int imageWidth, int imageHeight)
{
    // Code
    int xColumn = blockIdx.x * blockDim.x + threadIdx.x;
    int yRow = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (xColumn < imageWidth && yRow < imageHeight)
    {
        int pixelId = (yRow * imageWidth + xColumn) * 3;

        unsigned char pixelBlue = inputData[pixelId + 0];
        unsigned char pixelGreen = inputData[pixelId + 1];
        unsigned char pixelRed = inputData[pixelId + 2];

        float sepiaR = (
            (0.393f * pixelRed) + 
            (0.769f * pixelBlue) + 
            (0.189f * pixelGreen)
        );
        if (sepiaR > 255.0f) sepiaR = 255.0f;

        float sepiaG = (
            (0.349f * pixelRed) + 
            (0.686f * pixelBlue) + 
            (0.168f * pixelGreen)
        );
        if (sepiaG > 255.0f) sepiaG = 255.0f;
        
        float sepiaB = (
            (0.272f * pixelRed) + 
            (0.534f * pixelBlue) + 
            (0.131f * pixelGreen)
        );
        if (sepiaB > 255.0f) sepiaB = 255.0f;
        
        outputData[pixelId + 0] = static_cast<unsigned char>(sepiaB);
        outputData[pixelId + 1] = static_cast<unsigned char>(sepiaG);
        outputData[pixelId + 2] = static_cast<unsigned char>(sepiaR);
    }
}

__global__ void cudaColorInversion(unsigned char* inputData, unsigned char* outputData, int imageWidth, int imageHeight)
{
    // Code
    int xColumn = blockIdx.x * blockDim.x + threadIdx.x;
    int yRow = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (xColumn < imageWidth && yRow < imageHeight)
    {
        int pixelId = (yRow * imageWidth + xColumn) * 3;

        outputData[pixelId + 0] = 255 - inputData[pixelId + 0];
        outputData[pixelId + 1] = 255 - inputData[pixelId + 1];
        outputData[pixelId + 2] = 255 - inputData[pixelId + 2];
    }
}

__global__ void cudaGaussianBlur(unsigned char* inputData, unsigned char* outputData, float *kernel, int imageWidth, int imageHeight)
{
    // Code
    int xColumn = blockIdx.x * blockDim.x + threadIdx.x;
    int yRow = blockIdx.y * blockDim.y + threadIdx.y;

    if (xColumn < imageWidth && yRow < imageHeight)
    {
        int kernelRadius = CUDA_GAUSSIAN_KERNEL_SIZE / 2;
        float blurB = 0.0f, blurG = 0.0f, blurR = 0.0f;

        for (int i = -kernelRadius; i <= kernelRadius; i++)
        {
            for (int j = -kernelRadius; j <= kernelRadius; j++)
            {
                int xOffset = xColumn + i;
                int yOffset = yRow + j;

                if ((xOffset >= 0 && xOffset < imageWidth) && (yOffset >= 0 && yOffset < imageHeight))
                {
                   int pixelIndex = (yOffset * imageWidth + xOffset) * 3;
                   int kernelIndex = (i + kernelRadius) * CUDA_GAUSSIAN_KERNEL_SIZE + (j + kernelRadius);
                    
                   blurB += static_cast<float>(inputData[pixelIndex + 0]) * kernel[kernelIndex];
                   blurG += static_cast<float>(inputData[pixelIndex + 1]) * kernel[kernelIndex];
                   blurR += static_cast<float>(inputData[pixelIndex + 2]) * kernel[kernelIndex];
                }
            }
        }

        int outputIndex = (yRow * imageWidth + xColumn) * 3;
        outputData[outputIndex + 0] = static_cast<unsigned char>(blurB);
        outputData[outputIndex + 1] = static_cast<unsigned char>(blurG);
        outputData[outputIndex + 2] = static_cast<unsigned char>(blurR);
    }
}


void applyCUDAEffect(cv::Mat& image, int effectType, int& status)
{
    // Variable Declarations
    unsigned char* deviceInput = nullptr;
    unsigned char* deviceOutput = nullptr;

    //* Gaussian Blur Related
    float* hostKernel = nullptr;
    float* deviceKernel = nullptr;
    float kernelSum = 0.0f, sigma = 1.0f;
    int kernelSize;

    // Code
    int imageWidth = image.cols;
    int imageHeight = image.rows;
    size_t imagePixels = imageWidth * imageHeight;
    size_t dataSize = imagePixels * 3;

    CudaUtils *cudaUtils = new CudaUtils();

    if (!cudaUtils->memAlloc((void**)&deviceInput, dataSize))
        status = MEM_ALLOC_ERROR;

    if (!cudaUtils->memAlloc((void**)&deviceOutput, dataSize))
        status = MEM_ALLOC_ERROR;

    if (effectType == GAUSSIAN_BLUR_CUDA)
    {
        //* Create Gaussian Kernel
        hostKernel = new float[CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE];
        int kernelRadius = CUDA_GAUSSIAN_KERNEL_SIZE / 2;

        for (int i = -kernelRadius; i <= kernelRadius; i++) 
        {
            for (int j = -kernelRadius; j <= kernelRadius; j++)
            {
                int index = (i + kernelRadius) * kernelRadius + (j + kernelRadius);
                hostKernel[index] = exp(-(i * i + j + j) / (2.0f * sigma * sigma));
                kernelSum = kernelSum + hostKernel[index];
            }
        }
    
        for (int i = 0; i < CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE; i++)
            hostKernel[i] = hostKernel[i] / kernelSum;
        
        kernelSize = CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE * sizeof(float);
        
        if (!cudaUtils->memAlloc((void**)&deviceKernel, kernelSize))
            status = MEM_ALLOC_ERROR;
    }

    if (!cudaUtils->memCopy(deviceInput, image.data, dataSize, hipMemcpyHostToDevice))
        status = MEM_COPY_HOST_TO_DEVICE_ERROR;

    if (effectType == GAUSSIAN_BLUR_CUDA)
    {
        if (!cudaUtils->memCopy(deviceKernel, hostKernel, kernelSize, hipMemcpyHostToDevice))
            status = MEM_COPY_HOST_TO_DEVICE_ERROR;
    }

    //* CUDA Kernel Configuration
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(imageHeight, imageWidth);

    switch(effectType)
    {
        case DESATURATION_CUDA:
            cudaDesaturation<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, imageWidth, imageHeight);
        break;

        case SEPIA_CUDA:
            cudaSepia<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, imageWidth, imageHeight);
        break;

        case COLOR_INVERSION_CUDA:
            cudaColorInversion<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, imageWidth, imageHeight);
        break;

        case GAUSSIAN_BLUR_CUDA:
            cudaGaussianBlur<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, deviceKernel, imageWidth, imageHeight);
        break;
    }

    if (!cudaUtils->memCopy(image.data, deviceOutput, dataSize, hipMemcpyDeviceToHost))
        status = MEM_COPY_DEVICE_TO_HOST_ERROR;

    if (effectType == GAUSSIAN_BLUR_CUDA)
        cudaUtils->memFree((void**)&deviceKernel);

    cudaUtils->memFree((void**)&deviceOutput);
    cudaUtils->memFree((void**)&deviceInput);

    if (effectType == GAUSSIAN_BLUR_CUDA)
    {
        delete[] hostKernel;
        hostKernel = nullptr;
    }

    delete cudaUtils;
    cudaUtils = nullptr;

    status = SUCCESS;
}

int applyGaussianBlurCUDA(cv::Mat& image)
{
    // Variable Declarations
    float* hostKernel = nullptr;
    float* deviceKernel = nullptr;

    uchar* deviceInput = nullptr;
    uchar* deviceOutput = nullptr;

    float kernelSum = 0.0f, sigma = 1.0f;

    // Code

    //* Get Image Properties
    int imageWidth = image.cols;
    int imageHeight = image.rows;
    size_t imageSize = imageWidth * imageHeight * sizeof(uchar) * 3;

    //* Create Gaussian Kernel
    hostKernel = new float[CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE];
    int kernelRadius = CUDA_GAUSSIAN_KERNEL_SIZE / 2;

    for (int i = -kernelRadius; i <= kernelRadius; i++) 
    {
        for (int j = -kernelRadius; j <= kernelRadius; j++)
        {
            int index = (i + kernelRadius) * kernelRadius + (j + kernelRadius);
            hostKernel[index] = exp(-(i * i + j + j) / (2.0f * sigma * sigma));
            kernelSum = kernelSum + hostKernel[index];
        }
    }
   
    for (int i = 0; i < CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE; i++)
        hostKernel[i] = hostKernel[i] / kernelSum;
    
    int kernelSize = CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE * sizeof(float);

    CudaUtils *cudaUtils = new CudaUtils();

    

    // if (!cudaUtils->memAlloc((void**)&deviceInput, imageSize));
    //     return -1;
    // if (!cudaUtils->memAlloc((void**)&deviceOutput, imageSize));
    //     return -2;
    // if (!cudaUtils->memAlloc((void**)&deviceKernel, kernelSize));
    //     return -3;

    if (hipMalloc((void**)&deviceInput, imageSize) != hipSuccess)
        return -1;
    if (hipMalloc((void**)&deviceOutput, imageSize) != hipSuccess)
        return -2;
    if (hipMalloc((void**)&deviceKernel, kernelSize) != hipSuccess)
        return -3;

    // if (!cudaUtils->memCopy(deviceInput, image.data, imageSize, hipMemcpyHostToDevice))
    //     return -4;
    // if (!cudaUtils->memCopy(deviceKernel, hostKernel, kernelSize, hipMemcpyHostToDevice))
    //     return -5;

    if (hipMemcpy(deviceInput, image.data, imageSize, hipMemcpyHostToDevice) != hipSuccess)
        return -4;
    if (hipMemcpy(deviceKernel, hostKernel, kernelSize, hipMemcpyHostToDevice) != hipSuccess)
        return -5;

     // Kernel Configuration
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(imageHeight, imageWidth);

    cudaGaussianBlur<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, deviceKernel, imageWidth, imageHeight);
    hipDeviceSynchronize();

    // if (!cudaUtils->memCopy(image.data, deviceOutput, imageSize, hipMemcpyDeviceToHost))
    //     return -6;
    if (hipMemcpy(image.data, deviceOutput, imageSize, hipMemcpyDeviceToHost) != hipSuccess)
        return -6;

    cudaUtils->memFree((void**)&deviceKernel);
    cudaUtils->memFree((void**)&deviceOutput);
    cudaUtils->memFree((void**)&deviceInput);

    delete cudaUtils;
    cudaUtils = nullptr;

    delete[] hostKernel;
    hostKernel = nullptr;

    return 0;
}


