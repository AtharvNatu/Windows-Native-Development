#include "hip/hip_runtime.h"
#include "ImageEffects.cuh"

//! Cuda Utils Function Definitions
bool CudaUtils::memAlloc(void **devPtr, size_t size)
{
    // Code
    hipError_t result = hipMalloc(devPtr, size);
    if (result != hipSuccess)
        return false;
    return true;
}

bool CudaUtils::memSet(void *devPtr, int value, size_t count)
{
    // Code
    hipError_t result = hipMemset(devPtr, value, count);
    if (result != hipSuccess)
        return false;
    return true;
}

bool CudaUtils::memCopy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    // Code
    hipError_t result = hipMemcpy(dst, src, count, kind);
    if (result != hipSuccess)
        return false;
    return true;
}

void CudaUtils::memFree(void **devPtr)
{
    // Code
    if (*devPtr)
    {
        if (hipFree(*devPtr) == hipSuccess)
            *devPtr = nullptr;
    }
}

void CudaUtils::convertImageToPixelArr(uchar_t *imageData, uchar3 *pixelArray, size_t size)
{
    // Code
    for (size_t i = 0; i < size; i++, imageData += 3)
    {
        pixelArray[i].x = imageData[0];
        pixelArray[i].y = imageData[1];
        pixelArray[i].z = imageData[2];
    }
}

void CudaUtils::convertPixelArrToImage(uchar3 *pixelArray, uchar_t *imageData, size_t size)
{
    // Code
    for (size_t i = 0; i < size; i++, imageData += 3)
    {
        imageData[0] = pixelArray[i].x;
        imageData[1] = pixelArray[i].y;
        imageData[2] = pixelArray[i].z;
    }
}

__global__ void cudaDesaturation(uchar3* inputImageData, uchar3* outputImageData, size_t imageSize)
{
    // Code
    int pixelId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (pixelId < imageSize)
    {
        uchar grayscale = (uchar)(
                        (0.3f * (uchar)inputImageData[pixelId].z) +
                        (0.59f * (uchar)inputImageData[pixelId].y) +
                        (0.11f * (uchar)inputImageData[pixelId].x)
                    );
        
        outputImageData[pixelId].x = grayscale;
        outputImageData[pixelId].y = grayscale;
        outputImageData[pixelId].z = grayscale;
    }
}

__global__ void cudaDesaturation(unsigned char* inputImageData, unsigned char* outputImageData, size_t imageSize)
{
    // Code
    int pixelId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (pixelId < imageSize)
    {
        int idx = pixelId * 3;

        uchar grayscale = (uchar)(
                        (0.3f * (uchar)inputImageData[idx + 0]) +
                        (0.59f * (uchar)inputImageData[idx + 1]) +
                        (0.11f * (uchar)inputImageData[idx + 2])
                    );
        
        outputImageData[idx + 0] = grayscale;
        outputImageData[idx + 1] = grayscale;
        outputImageData[idx + 2] = grayscale;
    }
}

__global__ void cudaSepia(uchar3* inputImageData, uchar3* outputImageData, size_t imageSize)
{
    // Code
    int pixelId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (pixelId < imageSize)
    {
        float sepiaR = (
            (0.393f * static_cast<float>(inputImageData[pixelId].z)) + 
            (0.769f * static_cast<float>(inputImageData[pixelId].y)) + 
            (0.189f * static_cast<float>(inputImageData[pixelId].x))
        );
        if (sepiaR > 255.0f) sepiaR = 255.0f;

        float sepiaG = (
            (0.349f * static_cast<float>(inputImageData[pixelId].z)) + 
            (0.686f * static_cast<float>(inputImageData[pixelId].y)) + 
            (0.168f * static_cast<float>(inputImageData[pixelId].x))
        );
        if (sepiaG > 255.0f) sepiaG = 255.0f;
        
        float sepiaB = (
            (0.272f * static_cast<float>(inputImageData[pixelId].z)) + 
            (0.534f * static_cast<float>(inputImageData[pixelId].y)) + 
            (0.131f * static_cast<float>(inputImageData[pixelId].x))
        );
        if (sepiaB > 255.0f) sepiaB = 255.0f;
        
        outputImageData[pixelId].x = static_cast<uchar>(sepiaB);
        outputImageData[pixelId].y = static_cast<uchar>(sepiaG);
        outputImageData[pixelId].z = static_cast<uchar>(sepiaR);
    }
}

__global__ void cudaColorInversion(uchar3* inputImageData, uchar3* outputImageData, size_t imageSize)
{
    // Code
    int pixelId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (pixelId < imageSize)
    {
        outputImageData[pixelId].x = 255 - inputImageData[pixelId].x;
        outputImageData[pixelId].y = 255 - inputImageData[pixelId].y;
        outputImageData[pixelId].z = 255 - inputImageData[pixelId].z;
    }
}

__global__ void cudaGaussianBlur(unsigned char* inputData, unsigned char* outputData, int width, int height, float *kernel)
{
    // Code
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        float blur_pixel = 0.0f;
        int kernelRadius = CUDA_GAUSSIAN_KERNEL_SIZE / 2;

        for (int i = -kernelRadius; i <= kernelRadius; i++)
        {
            for (int j = -kernelRadius; j <= kernelRadius; j++)
            {
                int x_offset = x + i;
                int y_offset = y + j;

                if (x_offset >= 0 && x_offset < width && y_offset >= 0 && y_offset < height)
                {
                    int input_index = y_offset * width + x_offset;
                    int kernel_index = (i + kernelRadius) * CUDA_GAUSSIAN_KERNEL_SIZE + (j + kernelRadius);
                    blur_pixel = blur_pixel + static_cast<float>(inputData[input_index]) * kernel[kernel_index];
                }
            }
        }

        outputData[y * width + x] = static_cast<unsigned char>(blur_pixel);
    }
}


int applyDesaturationCUDA1(cv::Mat& image)
{
    uchar3* hostData = nullptr;
    uchar3* deviceInput = nullptr;
    uchar3* deviceOutput = nullptr;

    // Code
    size_t imageSize = image.size().height * image.size().width;

    hostData = new uchar3[imageSize];

    CudaUtils *cudaUtils = new CudaUtils();

    cudaUtils->convertImageToPixelArr(image.data, hostData, imageSize);

    if (!cudaUtils->memAlloc((void**)&deviceInput, imageSize * sizeof(uchar3)))
        return -1;

    if (!cudaUtils->memAlloc((void**)&deviceOutput, imageSize * sizeof(uchar3)))
        return -2;

    if (!cudaUtils->memCopy(deviceInput, hostData, imageSize * sizeof(uchar3), hipMemcpyHostToDevice))
        return -3;

    //* CUDA Kernel Configuration
    dim3 BLOCKS((imageSize + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK);

    cudaDesaturation<<<BLOCKS, THREADS_PER_BLOCK>>>(deviceInput, deviceOutput, imageSize);
    hipDeviceSynchronize();

    if (!cudaUtils->memCopy(hostData, deviceOutput, imageSize * sizeof(uchar3), hipMemcpyDeviceToHost))
        return -4;

    cudaUtils->convertPixelArrToImage(hostData, image.data, imageSize);

    cudaUtils->memFree((void**)&deviceOutput);
    cudaUtils->memFree((void**)&deviceInput);

    delete cudaUtils;
    cudaUtils = nullptr;

    delete[] hostData;
    hostData = nullptr;

    return 0;
}

int applyDesaturationCUDA(cv::Mat& image)
{
    unsigned char* deviceInput = nullptr;
    unsigned char* deviceOutput = nullptr;

    // Code
    size_t imagePixels = image.size().height * image.size().width;
    size_t dataSize = imagePixels * 3;

    CudaUtils *cudaUtils = new CudaUtils();

    if (!cudaUtils->memAlloc((void**)&deviceInput, dataSize))
        return -1;

    if (!cudaUtils->memAlloc((void**)&deviceOutput, dataSize))
        return -2;

    if (!cudaUtils->memCopy(deviceInput, image.data, dataSize, hipMemcpyHostToDevice))
        return -3;

    //* CUDA Kernel Configuration
    dim3 BLOCKS((imagePixels + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK);

    cudaDesaturation<<<BLOCKS, THREADS_PER_BLOCK>>>(deviceInput, deviceOutput, imagePixels);
    hipDeviceSynchronize();

    if (!cudaUtils->memCopy(image.data, deviceOutput, dataSize, hipMemcpyDeviceToHost))
        return -4;

    cudaUtils->memFree((void**)&deviceOutput);
    cudaUtils->memFree((void**)&deviceInput);

    delete cudaUtils;
    cudaUtils = nullptr;

    return 0;
}

int applySepiaCUDA(cv::Mat& image)
{
    uchar3* hostData = nullptr;
    uchar3* deviceInput = nullptr;
    uchar3* deviceOutput = nullptr;

    // Code
    size_t imageSize = image.size().height * image.size().width;

    hostData = new uchar3[imageSize];

    CudaUtils *cudaUtils = new CudaUtils();

    cudaUtils->convertImageToPixelArr(image.data, hostData, imageSize);

    if (!cudaUtils->memAlloc((void**)&deviceInput, imageSize * sizeof(uchar3)))
        return -1;

    if (!cudaUtils->memAlloc((void**)&deviceOutput, imageSize * sizeof(uchar3)))
        return -2;

    if (!cudaUtils->memCopy(deviceInput, hostData, imageSize * sizeof(uchar3), hipMemcpyHostToDevice))
        return -3;

    //* CUDA Kernel Configuration
    dim3 BLOCKS((imageSize + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK);

    cudaSepia<<<BLOCKS, THREADS_PER_BLOCK>>>(deviceInput, deviceOutput, imageSize);
    hipDeviceSynchronize();

    if (!cudaUtils->memCopy(hostData, deviceOutput, imageSize * sizeof(uchar3), hipMemcpyDeviceToHost))
        return -4;

    cudaUtils->convertPixelArrToImage(hostData, image.data, imageSize);

    cudaUtils->memFree((void**)&deviceOutput);
    cudaUtils->memFree((void**)&deviceInput);

    delete cudaUtils;
    cudaUtils = nullptr;

    delete[] hostData;
    hostData = nullptr;

    return 0;
}

int applyColorInversionCUDA(cv::Mat& image)
{
    uchar3* hostData = nullptr;
    uchar3* deviceInput = nullptr;
    uchar3* deviceOutput = nullptr;

    // Code
    size_t imageSize = image.size().height * image.size().width;

    hostData = new uchar3[imageSize];

    CudaUtils *cudaUtils = new CudaUtils();

    cudaUtils->convertImageToPixelArr(image.data, hostData, imageSize);

    if (!cudaUtils->memAlloc((void**)&deviceInput, imageSize * sizeof(uchar3)))
        return -1;

    if (!cudaUtils->memAlloc((void**)&deviceOutput, imageSize * sizeof(uchar3)))
        return -2;

    if (!cudaUtils->memCopy(deviceInput, hostData, imageSize * sizeof(uchar3), hipMemcpyHostToDevice))
        return -3;

    //* CUDA Kernel Configuration
    dim3 BLOCKS((imageSize + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK);

    cudaColorInversion<<<BLOCKS, THREADS_PER_BLOCK>>>(deviceInput, deviceOutput, imageSize);
    hipDeviceSynchronize();

    if (!cudaUtils->memCopy(hostData, deviceOutput, imageSize * sizeof(uchar3), hipMemcpyDeviceToHost))
        return -4;

    cudaUtils->convertPixelArrToImage(hostData, image.data, imageSize);

    cudaUtils->memFree((void**)&deviceOutput);
    cudaUtils->memFree((void**)&deviceInput);

    delete cudaUtils;
    cudaUtils = nullptr;

    delete[] hostData;
    hostData = nullptr;

    return 0;
}

int applyGaussianBlurCUDA(cv::Mat& image)
{
    // Variable Declarations
    float* hostKernel = nullptr;
    float* deviceKernel = nullptr;

    uchar* deviceInput = nullptr;
    uchar* deviceOutput = nullptr;

    float kernelSum = 0.0f, sigma = 1.0f;

    // Code

    //* Get Image Properties
    int imageWidth = image.cols;
    int imageHeight = image.rows;
    size_t imageSize = imageWidth * imageHeight * sizeof(uchar);

    //* Create Gaussian Kernel
    hostKernel = new float[CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE];
    int kernelRadius = CUDA_GAUSSIAN_KERNEL_SIZE / 2;

    for (int i = -kernelRadius; i <= kernelRadius; i++) 
    {
        for (int j = -kernelRadius; j <= kernelRadius; j++)
        {
            int index = (i + kernelRadius) * kernelRadius + (j + kernelRadius);
            hostKernel[index] = exp(-(i * i + j + j) / (2.0f * sigma * sigma));
            kernelSum = kernelSum + hostKernel[index];
        }
    }
   
    for (int i = 0; i < CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE; i++)
        hostKernel[i] = hostKernel[i] / kernelSum;
    
    int kernelSize = CUDA_GAUSSIAN_KERNEL_SIZE * CUDA_GAUSSIAN_KERNEL_SIZE * sizeof(float);

    CudaUtils *cudaUtils = new CudaUtils();

    

    // if (!cudaUtils->memAlloc((void**)&deviceInput, imageSize));
    //     return -1;
    // if (!cudaUtils->memAlloc((void**)&deviceOutput, imageSize));
    //     return -2;
    // if (!cudaUtils->memAlloc((void**)&deviceKernel, kernelSize));
    //     return -3;

    if (hipMalloc((void**)&deviceInput, imageSize) != hipSuccess)
        return -1;
    if (hipMalloc((void**)&deviceOutput, imageSize) != hipSuccess)
        return -2;
    if (hipMalloc((void**)&deviceKernel, kernelSize) != hipSuccess)
        return -3;

    // if (!cudaUtils->memCopy(deviceInput, image.data, imageSize, hipMemcpyHostToDevice))
    //     return -4;
    // if (!cudaUtils->memCopy(deviceKernel, hostKernel, kernelSize, hipMemcpyHostToDevice))
    //     return -5;

    if (hipMemcpy(deviceInput, image.data, imageSize, hipMemcpyHostToDevice) != hipSuccess)
        return -4;
    if (hipMemcpy(deviceKernel, hostKernel, kernelSize, hipMemcpyHostToDevice) != hipSuccess)
        return -5;

     // Kernel Configuration
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(imageHeight, imageWidth);

    cudaGaussianBlur<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, imageWidth, imageHeight, deviceKernel);
    hipDeviceSynchronize();

    // if (!cudaUtils->memCopy(image.data, deviceOutput, imageSize, hipMemcpyDeviceToHost))
    //     return -6;
    if (hipMemcpy(image.data, deviceOutput, imageSize, hipMemcpyDeviceToHost) != hipSuccess)
        return -6;

    cudaUtils->memFree((void**)&deviceKernel);
    cudaUtils->memFree((void**)&deviceOutput);
    cudaUtils->memFree((void**)&deviceInput);

    delete cudaUtils;
    cudaUtils = nullptr;

    delete[] hostKernel;
    hostKernel = nullptr;

    return 0;
}


