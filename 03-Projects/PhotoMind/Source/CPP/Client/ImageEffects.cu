#include "hip/hip_runtime.h"
#include "ImageEffects.cuh"

//! Cuda Utils Function Definitions
bool CudaUtils::memAlloc(void **devPtr, size_t size)
{
    // Code
    hipError_t result = hipMalloc(devPtr, size);
    if (result != hipSuccess)
        return false;
    return true;
}

bool CudaUtils::memSet(void *devPtr, int value, size_t count)
{
    // Code
    hipError_t result = hipMemset(devPtr, value, count);
    if (result != hipSuccess)
        return false;
    return true;
}

bool CudaUtils::memCopy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    // Code
    hipError_t result = hipMemcpy(dst, src, count, kind);
    if (result != hipSuccess)
        return false;
    return true;
}

void CudaUtils::memFree(void **devPtr)
{
    // Code
    if (*devPtr)
    {
        if (hipFree(*devPtr) == hipSuccess)
            *devPtr = nullptr;
    }
}

void CudaUtils::convertImageToPixelArr(uchar_t *imageData, uchar3 *pixelArray, size_t size)
{
    // Code
    for (size_t i = 0; i < size; i++, imageData += 3)
    {
        pixelArray[i].x = imageData[0];
        pixelArray[i].y = imageData[1];
        pixelArray[i].z = imageData[2];
    }
}

void CudaUtils::convertPixelArrToImage(uchar3 *pixelArray, uchar_t *imageData, size_t size)
{
    // Code
    for (size_t i = 0; i < size; i++, imageData += 3)
    {
        imageData[0] = pixelArray[i].x;
        imageData[1] = pixelArray[i].y;
        imageData[2] = pixelArray[i].z;
    }
}

__global__ void cudaDesaturation(uchar3* inputImageData, uchar3* outputImageData, size_t imageSize)
{
    // Code
    int pixelId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (pixelId < imageSize)
    {
        uchar grayscale = (uchar)(
                        (0.3f * (uchar)inputImageData[pixelId].z) +
                        (0.59f * (uchar)inputImageData[pixelId].y) +
                        (0.11f * (uchar)inputImageData[pixelId].x)
                    );
        
        outputImageData[pixelId].x = grayscale;
        outputImageData[pixelId].y = grayscale;
        outputImageData[pixelId].z = grayscale;
    }
}

#include <fstream>

void logMessage(const std::string& message) {
    FILE* logFile = fopen("log.txt", "a+");
    if (logFile) {
        fprintf(logFile, "%s\n", message.c_str());
        fclose(logFile);
    }
}

// void applyDesatCUDA(cv::Mat& image)
// {
//     uchar3* hostImage = nullptr;
//     uchar3* deviceInput = nullptr;
//     uchar3* deviceOutput = nullptr;

//     size_t imageSize = image.size().height * image.size().width;
//     hostImage = new uchar3[imageSize];

//     CudaUtils *cudaUtils = new CudaUtils();

//     bool isBGR = false;
//     if (image.channels() == 3 && image.type() == CV_8UC3) {
//         isBGR = true;
//     }

//     logMessage("Checking image format...");
//     if (isBGR) {
//         logMessage("Image is in BGR format. Proceeding with conversion.");
//         cudaUtils->convertImageToPixelArr(image.data, hostImage, imageSize);
//     } else {
//         logMessage("Image is not in BGR format. Converting...");
//         cv::Mat tempImage;
//         cv::cvtColor(image, tempImage, cv::COLOR_RGB2BGR);
//         cudaUtils->convertImageToPixelArr(tempImage.data, hostImage, imageSize);
//     }

//     if (!hipMalloc((void**)&deviceInput, imageSize * sizeof(uchar3)) ||
//         !hipMalloc((void**)&deviceOutput, imageSize * sizeof(uchar3))) {
//         logMessage("CUDA memory allocation failed!");
//         return;
//     }
//     logMessage("Memory allocated successfully.");

//     if (!hipMemcpy(deviceInput, hostImage, imageSize * sizeof(uchar3), hipMemcpyHostToDevice)) {
//         logMessage("CUDA memory copy to device failed!");
//         return;
//     }
//     logMessage("Memory copied to device successfully.");

//     dim3 BLOCKS((imageSize + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK);
//     logMessage("Launching CUDA kernel...");

//     cudaDesaturation<<<BLOCKS, THREADS_PER_BLOCK>>>(deviceInput, deviceOutput, imageSize);
//     hipError_t err = hipGetLastError();
//     if (err != hipSuccess) {
//         logMessage("Kernel launch failed: " + std::string(hipGetErrorString(err)));
//         return;
//     }

//     hipDeviceSynchronize();
//     logMessage("CUDA kernel execution completed.");

//     hipError_t err1 = hipMemcpy(deviceOutput, hostImage, imageSize * sizeof(uchar3), hipMemcpyDeviceToHost);
//     if (err1 != hipSuccess) {
//         logMessage("CUDA memory copy from device failed! Error: " + std::string(hipGetErrorString(err)));
//         return;
//     }
//     logMessage("Memory copied from device successfully.");

//     cudaUtils->convertPixelArrToImage(hostImage, image.data, imageSize);

//     if (!cv::imwrite("output_desaturated.png", image)) {
//         logMessage("Error writing the image!");
//         return;
//     }
//     logMessage("Desaturated image saved successfully.");

//     cudaUtils->memFree((void**)&deviceOutput);
//     cudaUtils->memFree((void**)&deviceInput);

//     delete cudaUtils;
//     delete[] hostImage;
// }


int applyDesatCUDA(cv::Mat& image)
{
    uchar3* hostImage = nullptr;
    uchar3* outputData = nullptr;
    uchar3* deviceInput = nullptr;
    uchar3* deviceOutput = nullptr;
    hipError_t result;

    size_t imageSize = image.size().height * image.size().width;

    hostImage = new uchar3[imageSize];
    outputData = new uchar3[imageSize];

    CudaUtils *cudaUtils = new CudaUtils();

    cudaUtils->convertImageToPixelArr(image.data, hostImage, imageSize);

    if (hipMalloc((void**)&deviceInput, imageSize * sizeof(uchar3)) != hipSuccess)
        return -1;

    if (hipMalloc((void**)&deviceOutput, imageSize * sizeof(uchar3)) != hipSuccess)
        return -2;

    if (hipMemcpy(deviceInput, hostImage, imageSize * sizeof(uchar3), hipMemcpyHostToDevice) != hipSuccess)
        return -3;


    //* CUDA Kernel Configuration
    dim3 BLOCKS((imageSize + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK);

    cudaDesaturation<<<BLOCKS, THREADS_PER_BLOCK>>>(deviceInput, deviceOutput, imageSize);
    hipDeviceSynchronize();

    if (hipMemcpy(outputData, deviceOutput, imageSize * sizeof(uchar3), hipMemcpyDeviceToHost) != hipSuccess)
        return -4;

    cudaUtils->convertPixelArrToImage(outputData, image.data, imageSize);

    hipFree((void**)&deviceOutput);
    hipFree((void**)&deviceInput);

    delete cudaUtils;
    cudaUtils = nullptr;

    delete[] outputData;
    outputData = nullptr;

    delete[] hostImage;
    hostImage = nullptr;

    return 0;
}

